// STD
#include <iostream>
#include <filesystem>
#include <string>
#include <vector>

// Boost
#include <boost/program_options.hpp>

#include "file_utils.cuh"
#include "kernels.cuh"

static std::filesystem::path output_path;

void convert_image(const char *filepath) {

	std::string output_name = output_path / std::filesystem::path(filepath).stem();

/* Load image */
  unsigned char *d_input_image = nullptr;
  int width, height, channels;
  if (!read_texture_to_device(d_input_image, filepath, &width, &height, &channels))
    return;

  size_t size = width * height;


  // TODO delete
  write_device_texture_to_file((output_name + ".png").c_str(), d_input_image, width, height, 1);

  // Threads/blocks
  // TODO what's optimal?
  dim3 threads(16, 16);
  dim3 blocks((width + threads.x - 1) / threads.x,
              (height + threads.y - 1) / threads.y);

/* First order derivatives */
  // Allocate device memory
  int *d_fod;
  unsigned char *d_fod_image, *d_fod_dirs, *d_fod_dirs_image;

  CUDA_CHECK(hipMalloc(&d_fod, size * 2 * sizeof(int)));
  CUDA_CHECK(hipMalloc(&d_fod_image, size * 3));
  CUDA_CHECK(hipMalloc(&d_fod_dirs, size));
  CUDA_CHECK(hipMalloc(&d_fod_dirs_image, size));

  // Launch kernel
  first_derivative<<<blocks, threads>>>(d_input_image, d_fod_image, d_fod_dirs,
                                        d_fod_dirs_image, d_fod, width, height);
  CUDA_CHECK(hipDeviceSynchronize());

  // Write result image to file
  write_device_texture_to_file((output_name + "_fod.png").c_str(), d_fod_image,
                               width, height, 3);
  write_device_texture_to_file((output_name + "_fod_dirs.png").c_str(),
                               d_fod_dirs_image, width, height, 1);

/* Directional local maxima */
  // Allocate device memory
  unsigned char *d_local_max_8dirs, *d_dir_bit_image;
  CUDA_CHECK(hipMalloc(&d_local_max_8dirs, size));
  CUDA_CHECK(hipMalloc(&d_dir_bit_image, size));

  // Launch kernel
  local_max_8dirs<<<blocks, threads>>>(d_input_image, d_local_max_8dirs,
																			 width, height);
  CUDA_CHECK(hipDeviceSynchronize());

	//  // Save local maxima in each direction to separate images
	// for (int i = 0; i < 8; ++i) {
	// 	bits_to_image<<<blocks, threads>>>(d_local_max_8dirs, d_dir_bit_image,
	// 																		width, height, 1 << i);
	// 	CUDA_CHECK(hipDeviceSynchronize());
	// 	write_device_texture_to_file((output_name + "_local_max_dir" + std::to_string(i) + ".png").c_str(),
	// 															 d_dir_bit_image, width, height, 1);
	// }

	// Any of the 8
	bits_to_image<<<blocks, threads>>>(d_local_max_8dirs, d_dir_bit_image,
																		width, height, 0b11111111);
  CUDA_CHECK(hipDeviceSynchronize());
	write_device_texture_to_file((output_name + "_local_max_8dirs.png").c_str(),
															 d_dir_bit_image, width, height, 1);

	// Any of the 4 axis aligned dirs
	bits_to_image<<<blocks, threads>>>(d_local_max_8dirs, d_dir_bit_image,
																		width, height, 0b01010101);
  CUDA_CHECK(hipDeviceSynchronize());
	write_device_texture_to_file((output_name + "_local_max_4dirs.png").c_str(),
															 d_dir_bit_image, width, height, 1);


// /* Create binary mipmaps */
//   int mipmap_width  = (width  + 1) / 2;
//   int mipmap_height = (height + 1) / 2;
//
//   unsigned char *d_binary_mipmap;
//   CUDA_CHECK(hipMalloc(&d_binary_mipmap, mipmap_width * mipmap_height));
//
// 	create_binary_mipmap_level<<<blocks, threads>>>(d_binary_mipmap, d_local_max_8dirs, width, height, mipmap_width, mipmap_height);
// 	CUDA_CHECK(hipDeviceSynchronize());
//
//   CUDA_CHECK(hipFree(d_dir_bit_image));
//   CUDA_CHECK(hipMalloc(&d_dir_bit_image, mipmap_width * mipmap_height));
//
// 	// Any of the 8
// 	bits_to_image<<<blocks, threads>>>(d_binary_mipmap, d_dir_bit_image,
// 																		mipmap_width, mipmap_height, 0b11111111);
//   CUDA_CHECK(hipDeviceSynchronize());
// 	write_device_texture_to_file((output_name + "_local_max_8dirs_mipmap.png").c_str(),
// 															 d_dir_bit_image, mipmap_width, mipmap_height, 1);
//
// 	// Any of the 4 axis aligned dirs
// 	bits_to_image<<<blocks, threads>>>(d_binary_mipmap, d_dir_bit_image,
// 																		mipmap_width, mipmap_height, 0b01010101);
//   CUDA_CHECK(hipDeviceSynchronize());
// 	write_device_texture_to_file((output_name + "_local_max_4dirs_mipmap.png").c_str(),
// 															 d_dir_bit_image, mipmap_width, mipmap_height, 1);

/* Relaxed cone map generation */
  // Allocate device memory
  unsigned char *d_cone_map;
  CUDA_CHECK(hipMalloc(&d_cone_map, size * 4));

  // Launch kernel
  create_cone_map_8dirs<<<blocks, threads>>>(d_input_image, d_fod_image,
																						 d_local_max_8dirs, d_cone_map,
																						 width, height);
  CUDA_CHECK(hipDeviceSynchronize());

  // Write result image to file
  write_device_texture_to_file((output_name + "_relaxed_cone_map_8dirs.png").c_str(),
                               d_cone_map, width, height, 4);

  // Launch kernel
  create_cone_map_4dirs<<<blocks, threads>>>(d_input_image, d_fod_image,
																						 d_local_max_8dirs, d_cone_map,
																						 width, height);
  CUDA_CHECK(hipDeviceSynchronize());

  // Write result image to file
  write_device_texture_to_file((output_name + "_relaxed_cone_map_4dirs.png").c_str(),
                               d_cone_map, width, height, 4);


/* Second order derivatives and watershed */
  // Allocate device memory
  unsigned char *d_sod_image, *d_watershed;
  CUDA_CHECK(hipMalloc(&d_sod_image, size * 4));
  CUDA_CHECK(hipMalloc(&d_watershed, size));

  // Launch kernel
  second_derivative<<<blocks, threads>>>(d_fod, d_sod_image, d_watershed,
																				 width, height);
  CUDA_CHECK(hipDeviceSynchronize());

  // Write result image to file
  write_device_texture_to_file((output_name + "_sod.png").c_str(), d_sod_image,
                               width, height, 4);
  write_device_texture_to_file((output_name + "_watershed.png").c_str(), d_watershed,
                               width, height, 1);

/* Non maximum suppression */
  // Allocate device memory
  unsigned char *d_suppressed;
  CUDA_CHECK(hipMalloc(&d_suppressed, size));

  // Launch kernel
  non_maximum_suppression<<<blocks, threads>>>(d_input_image, d_fod_dirs,
                                               d_watershed, d_suppressed,
                                               width, height);
  CUDA_CHECK(hipDeviceSynchronize());

  // Write result image to file
  write_device_texture_to_file((output_name + "_suppressed.png").c_str(),
                               d_suppressed, width, height, 1);


/* Relaxed cone map generation */
  // Launch kernel
  create_cone_map_analytic<<<blocks, threads>>>(d_input_image, d_fod_image, d_fod_dirs, d_suppressed,
																								d_cone_map, width, height);
  CUDA_CHECK(hipDeviceSynchronize());

  // Write result image to file
  write_device_texture_to_file((output_name + "_relaxed_cone_map_analytic.png").c_str(),
                               d_cone_map, width, height, 4);

/* Cleanup */
  CUDA_CHECK(hipFree(d_input_image));
  CUDA_CHECK(hipFree(d_fod));
  CUDA_CHECK(hipFree(d_fod_image));
  CUDA_CHECK(hipFree(d_fod_dirs));
  CUDA_CHECK(hipFree(d_fod_dirs_image));
  CUDA_CHECK(hipFree(d_sod_image));
  CUDA_CHECK(hipFree(d_watershed));
  CUDA_CHECK(hipFree(d_suppressed));
  CUDA_CHECK(hipFree(d_local_max_8dirs));
  CUDA_CHECK(hipFree(d_dir_bit_image));
  // CUDA_CHECK(hipFree(d_binary_mipmap));
  CUDA_CHECK(hipFree(d_cone_map));
}

int main(int argc, char* argv[]) {
  std::vector<std::string> input_files;

  // Possible options
  boost::program_options::options_description desc("Allowed options");
  desc.add_options()
    ("help,h", "produce help message")
    ("output,o", boost::program_options::value<std::filesystem::path>(&output_path)->default_value("."), "set path to output folder")
    ("input,i", boost::program_options::value<std::vector<std::string>>(&input_files), "input files");
		//TODO flip Y, depthmap, wrap

  // Positional options
  boost::program_options::positional_options_description pod;
  pod.add("input", -1);  // all remaining options

  boost::program_options::variables_map vm;

  try {
    boost::program_options::store(boost::program_options::command_line_parser(argc, argv).options(desc).positional(pod).run(), vm);
    boost::program_options::notify(vm);
  } catch (std::exception& e) {
    std::cerr << "Error: " << e.what() << "\n";
    return 1;
  }

	// Help
  if (vm.count("help")) {
    std::cout << "Usage: " << argv[0] << " [-o OUTPUT] INPUT [INPUT]...\n";
    std::cout << desc << "\n";
    return 0;
  }

	// No input
  if (input_files.empty()) {
    std::cerr << "Error: No input files provided.\n";
    return 1;
  }

  // Output
  if (!std::filesystem::exists(output_path)) {
		std::filesystem::create_directory(output_path);
  } else if (!std::filesystem::is_directory(output_path)) {
		std::cerr << "Error: " << output_path << " is not a directory.\n";
		return 1;
  }

	// OK
  for (auto file : input_files) {
    if (!std::filesystem::exists(file) || !std::filesystem::is_regular_file(file)) {
			std::cerr << "Error: " << file << " is not a file.\n";
			continue;
    }
    convert_image(file.c_str());
  }

  return 0;
}
