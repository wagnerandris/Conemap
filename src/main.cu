#include "hip/hip_runtime.h"
// STB
#include <cmath>
#include <cstdlib>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb/stb_image.h>
#include <stb/stb_image_write.h>

// STD
#include <stdio.h>
#include <filesystem>
#include <string>

// CUDA
#include <hip/hip_runtime.h>

// CUDA error check macro
#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line)
{
  if (code != hipSuccess)
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);
  }
}

// TODO is Sobel better than Prewitt?
__device__ __constant__ float vkernel[9] = {
 -1,-2,-1,
	0, 0, 0,
	1, 2, 1
};

__device__ __constant__ float hkernel[9] = {
 -1, 0, 1,
 -2, 0, 2,
 -1, 0, 1
};

__global__ void first_derivative(unsigned char* heightmap, unsigned char* derivative_image, unsigned char* dirs, unsigned char* dirs_image, int* fod, int width, int height, int channels) {
  int u = blockIdx.x * blockDim.x + threadIdx.x;
  int v = blockIdx.y * blockDim.y + threadIdx.y;

  if (u >= width || v >= height) return;
	
  // Possible optimizations:
  //  - For every pixel, compute the sum with the side adjacent ones in both dimensions and save to shared memory
	//  - Step by channels instead of 1 in the loop (only worth it, if kernel lookup is independent)
	//  - Previtt operator instead of Sobel -> no kernel needed multiply by value/sign of index
	int hsum = 0;
	int vsum = 0;
	for (int dv = 0; dv < 3; ++dv) {
		for (int du = 0; du < 3; ++du) {
      int cu = min(max(u + du - 1, 0), width - 1);
      int cv = min(max(v + dv - 1, 0), height - 1);
      int cidx = (cv * width + cu) * channels;
			hsum += heightmap[cidx] * hkernel[dv * 3 + du];
			vsum += heightmap[cidx] * vkernel[dv * 3 + du];
		}
	}

	// int grad[2] = {hsum, vsum};
	// int scale = std::sqrt(vsum * vsum + hsum * hsum);
	float dir = atan2f(vsum, hsum);
	unsigned char discrete_dir = static_cast<unsigned char>(
																(dir + M_PI // all positive
																+ M_PI_4f / 2.0f) // align regions
																/ M_PI_4f // 8 dirs
																) % 4; // the 4 we care about;

	int idx = v * width + u;

	derivative_image[idx * 3 + 0] = (hsum + 1020) >> 3;
	derivative_image[idx * 3 + 1] = (vsum + 1020) >> 3;
	derivative_image[idx * 3 + 2] = 127;

	fod[idx * 2 + 0] = hsum;
	fod[idx * 2 + 1] = vsum;

	dirs[v * width + u] = discrete_dir;

	dirs_image[v * width + u] = dirs[v * width + u] * 32 + 127;
}

__global__ void second_derivative(int* fod, unsigned char* sod_image, unsigned char* watershed_image, int width, int height) {
  int u = blockIdx.x * blockDim.x + threadIdx.x;
  int v = blockIdx.y * blockDim.y + threadIdx.y;

  if (u >= width || v >= height) return;
	
  // Possible optimizations:
  //  - For every pixel, compute the sum with the side adjacent ones in both dimensions and save to shared memory
	//  - Step by channels instead of 1 in the loop (only worth it, if kernel lookup is independent)
	//  - Previtt operator instead of Sobel -> no kernel needed multiply by value/sign of index
	int hhsum = 0;
	int vhsum = 0;
	int hvsum = 0;
	int vvsum = 0;
	for (int dv = 0; dv < 3; ++dv) {
		for (int du = 0; du < 3; ++du) {
      int cu = min(max(u + du - 1, 0), width - 1);
      int cv = min(max(v + dv - 1, 0), height - 1);
      int cidx = (cv * width + cu) * 2;
			hhsum += fod[cidx + 0] * hkernel[dv * 3 + du];
			hvsum += fod[cidx + 1] * hkernel[dv * 3 + du];
			vhsum += fod[cidx + 0] * vkernel[dv * 3 + du];
			vvsum += fod[cidx + 1] * vkernel[dv * 3 + du];
		}
	}

	int idx = v * width + u;

	sod_image[idx * 4 + 0] = (hhsum + 1020) >> 3;
	sod_image[idx * 4 + 1] = (hvsum + 1020) >> 3;
	sod_image[idx * 4 + 2] = (vhsum + 1020) >> 3;
	sod_image[idx * 4 + 3] = (vvsum + 1020) >> 3;
	
	int val = hhsum * fod[idx * 2 + 1] * fod[idx * 2 + 1] - (hvsum + vhsum) * fod[idx * 2 + 0] * fod[idx * 2 + 1] + vvsum * fod[idx * 2 + 0] * fod[idx * 2 + 0];

	watershed_image[idx] = val < 0 ? 255 : 0;
}

__global__ void non_maximum_suppression(unsigned char* heightmap, unsigned char* dirs, unsigned char* watershed_image, unsigned char* suppressed_image, int width, int height, int channels) {
  int u = blockIdx.x * blockDim.x + threadIdx.x;
  int v = blockIdx.y * blockDim.y + threadIdx.y;

  if (u >= width || v >= height) return;

	int du;
	int dv;

	switch (dirs[v * width + u]) {
		case 0:
			du = 0;
			dv = 1;
			break;
		case 1:
			du = -1;
			dv = 1;
			break;
		case 2:
			du = 1;
			dv = 0;
			break;
		case 3:
			du = 1;
			dv = 1;
			break;
	}

	if (watershed_image[v * width + u] == 0) {
		suppressed_image[v * width + u] = 0;
	} else
	if (u + du >= 0 && u + du < width &&
			v + dv >= 0 && v + dv < height &&
			heightmap[((v + dv) * width + u + du) * channels] > heightmap[(v * width + u) * channels]) {
			suppressed_image[v * width + u] = 0;
	} else
	if (u - du >= 0 && u - du < width &&
			v - dv >= 0 && v - dv < height &&
			heightmap[((v - dv) * width + u - du) * channels] > heightmap[(v * width + u) * channels]) {
			suppressed_image[v * width + u] = 0;
	} else {
		suppressed_image[v * width + u] = 255;
	}
}

__global__ void create_cone_map(unsigned char* heightmap, unsigned char* derivative_image, unsigned char* suppressed_image, unsigned char* cone_map, int width, int height, int channels) {
  int u = blockIdx.x * blockDim.x + threadIdx.x;
  int v = blockIdx.y * blockDim.y + threadIdx.y;

  if (u >= width || v >= height) return;

	float iwidth = 1.0f / width;
	float iheight = 1.0f / height;

	// TODO Why are we assuming run/rise = 1, instead of infinity?
	float min_ratio2 = 1.0f;

	// normalize height
	float h = heightmap[(v * width + u) * channels] / 255.0f;

	// init variables
	int du, dv;
	float dun, dvn;
	int start, end;

	// search in an increasing radius spiral around the texel
	for (int rad = 1;
			rad * rad <= 1.1f * (1.0f - h) * width *
									 1.1f * (1.0f - h) * height *
									 min_ratio2 &&
			rad <= 1.1f * (1.0f - h) * width &&
			rad <= 1.1f * (1.0f - h) * height;
			++rad) {

		// Left side

		// u displacement	
		du = u - rad;
		// normalized
		dun = -rad * iwidth;

		// TODO only if tileable option is set
		// loop around until reaching valid coordinates
		while (du < 0) du += width; 
		// TODO symmetrical corners
		// set v limits
		start = max(v - rad + 1, 0);
		end = min(v + rad - 1, height);

		// go through side
		for (int dv = start; dv < end; ++dv) {
			// TODO check if (suppressed) watershed point, skip if not
			// normalize v displacement
			dvn = (dv - v) * iheight;

			// distance squared
			float d2 = dun * dun + dvn * dvn;

			// height difference
			float dh = heightmap[(dv * width + du) * channels] / 255.0 - h;

			// if more steep than previous best, override
			if (dh > 0.0f && dh * dh * min_ratio2 > d2) {
				min_ratio2 = d2 / (dh * dh);
			}
		}

		// Right side

		// u displacement	
		du = u + rad;
		// normalized
		dun = rad * iwidth;

		// TODO only if tileable option is set
		// loop around until reaching valid coordinates
		while (du >= width) du -= width; 
		// TODO symmetrical corners
		// set v limits
		start = max(v - rad + 1, 0);
		end = min(v + rad - 1, height);

		// go through side
		for (int dv = start; dv < end; ++dv) {
			// TODO check if (suppressed) watershed point, skip if not
			// normalize v displacement
			dvn = (dv - v) * iheight;

			// distance squared
			float d2 = dun * dun + dvn * dvn;

			// height difference
			float dh = heightmap[(dv * width + du) * channels] / 255.0 - h;

			// if more steep than previous best, override
			if (dh > 0.0f && dh * dh * min_ratio2 > d2) {
				min_ratio2 = d2 / (dh * dh);
			}
		}

		// Top side

		// u displacement	
		dv = v - rad;
		// normalized
		dvn = -rad * iheight;

		// TODO only if tileable option is set
		// loop around until reaching valid coordinates
		while (dv < 0) dv += height; 
		// TODO symmetrical corners
		// set u limits
		start = max(u - rad + 1, 0);
		end = min(u + rad - 1, width);

		// go through side
		for (int du = start; du < end; ++du) {
			// TODO check if (suppressed) watershed point, skip if not
			// normalize v displacement
			dun = (du - u) * iwidth;

			// distance squared
			float d2 = dun * dun + dvn * dvn;

			// height difference
			float dh = heightmap[(dv * width + du) * channels] / 255.0 - h;

			// if more steep than previous best, override
			if (dh > 0.0f && dh * dh * min_ratio2 > d2) {
				min_ratio2 = d2 / (dh * dh);
			}
		}

		// Bottom side

		// u displacement	
		dv = v + rad;
		// normalized
		dvn = rad * iheight;

		// TODO only if tileable option is set
		// loop around until reaching valid coordinates
		while (dv >= height) dv -= height; 
		// TODO symmetrical corners
		// set u limits
		start = max(u - rad + 1, 0);
		end = min(u + rad - 1, width);

		// go through side
		for (int du = start; du < end; ++du) {
			// TODO check if (suppressed) watershed point, skip if not
			// normalize v displacement
			dun = (du - u) * iwidth;

			// distance squared
			float d2 = dun * dun + dvn * dvn;

			// height difference
			float dh = heightmap[(dv * width + du) * channels] / 255.0 - h;

			// if more steep than previous best, override
			if (dh > 0.0f && dh * dh * min_ratio2 > d2) {
				min_ratio2 = d2 / (dh * dh);
			}
		}
	}

	float ratio = sqrt(min_ratio2);
  // most of the data is on the low end...sqrting again spreads it better
  // (plus multiply is a cheap operation in shaders!)
  // -- Dummer
	ratio = sqrt(ratio);
	cone_map[(v * width + u) * 4 + 0] = heightmap[(v * width + u) * channels];
	cone_map[(v * width + u) * 4 + 1] = static_cast<unsigned char>(ratio * 255);
	cone_map[(v * width + u) * 4 + 2] = derivative_image[(v * width + u) * 3];
	cone_map[(v * width + u) * 4 + 3] = derivative_image[(v * width + u) * 3 + 1];
}

unsigned char* read_texture_from_file(const char* filepath, int* w, int* h, int* c) {
  unsigned char* data = stbi_load(filepath, w, h, c, 0);
  if (!data) {
    fprintf(stderr, "Could not load texture from %s.\n", filepath);
    return nullptr;
  }
  printf("Loaded texture from %s.\nWidth: %d, Height: %d, Channels: %d\n", filepath, *w, *h, *c);
  return data;
}

void convert_image(const char* filepath) {
  int width, height, channels;

// Load image
  unsigned char* h_input = read_texture_from_file(filepath, &width, &height, &channels);
  if (!h_input) return;

// First order derivatives
  size_t size = width * height;
  size_t input_size = size * channels;
  unsigned char *d_input, *d_derivative_image, *d_dirs, *d_dirs_image;
  int* d_fod;

  // Allocate device memory
  CUDA_CHECK(hipMalloc(&d_input, input_size));
  CUDA_CHECK(hipMalloc(&d_derivative_image, size * 3));
  CUDA_CHECK(hipMalloc(&d_dirs, size));
  CUDA_CHECK(hipMalloc(&d_dirs_image, size));
  CUDA_CHECK(hipMalloc(&d_fod, size * 2 * sizeof(int)));

  // Copy image to GPU
  CUDA_CHECK(hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice));

  // Launch kernel
  dim3 threads(16, 16);
  dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
  first_derivative<<<blocks, threads>>>(d_input, d_derivative_image, d_dirs, d_dirs_image, d_fod, width, height, channels);
  CUDA_CHECK(hipDeviceSynchronize());

  // Copy result back
  unsigned char* h_derivative_image = (unsigned char*)malloc(size * 3);
  CUDA_CHECK(hipMemcpy(h_derivative_image, d_derivative_image, size * 3, hipMemcpyDeviceToHost));
  unsigned char* h_dirs_image = (unsigned char*)malloc(size);
  CUDA_CHECK(hipMemcpy(h_dirs_image, d_dirs_image, size, hipMemcpyDeviceToHost));

  // Save image
	std::string derivative_file = std::filesystem::path(filepath).stem().string() + "_derivative.png";
  stbi_write_png(derivative_file.c_str(), width, height, 3, h_derivative_image, width * 3);
	printf("Written image as %s\n", derivative_file.c_str());
	
	std::string dirs_image_file = std::filesystem::path(filepath).stem().string() + "_dirs_image.png";
  stbi_write_png(dirs_image_file.c_str(), width, height, 1, h_dirs_image, width * 1);
  printf("Written image as %s\n", dirs_image_file.c_str());

// Second order derivatives and watershed
	unsigned char *d_second_derivative_image, *d_watershed;

  CUDA_CHECK(hipMalloc(&d_second_derivative_image, size * 4));
  CUDA_CHECK(hipMalloc(&d_watershed, size));
  
  // Launch kernel
  second_derivative<<<blocks, threads>>>(d_fod, d_second_derivative_image, d_watershed, width, height);
  CUDA_CHECK(hipDeviceSynchronize());

  // Copy result back
  unsigned char* h_second_derivative_image = (unsigned char*)malloc(size * 4);
  CUDA_CHECK(hipMemcpy(h_second_derivative_image, d_second_derivative_image, size * 4, hipMemcpyDeviceToHost));
  unsigned char* h_watershed = (unsigned char*)malloc(size);
  CUDA_CHECK(hipMemcpy(h_watershed, d_watershed, size, hipMemcpyDeviceToHost));

  // Save image
	std::string second_derivative_file = std::filesystem::path(filepath).stem().string() + "_second_derivative.png";
  stbi_write_png(second_derivative_file.c_str(), width, height, 4, h_second_derivative_image, width * 4);
  printf("Written image as %s\n", second_derivative_file.c_str());
	
	std::string watershed_file = std::filesystem::path(filepath).stem().string() + "_watershed.png";
  stbi_write_png(watershed_file.c_str(), width, height, 1, h_watershed, width * 1);
  printf("Written image as %s\n", watershed_file.c_str());

// Non maximum suppression
	unsigned char *d_suppressed;

  CUDA_CHECK(hipMalloc(&d_suppressed, size));
  
  // Launch kernel
  non_maximum_suppression<<<blocks, threads>>>(d_input, d_dirs, d_watershed, d_suppressed, width, height, channels);
  CUDA_CHECK(hipDeviceSynchronize());
  
  // Copy result back
  unsigned char* h_suppressed = (unsigned char*)malloc(size);
  CUDA_CHECK(hipMemcpy(h_suppressed, d_suppressed, size, hipMemcpyDeviceToHost));

  // Save image
	std::string suppressed_file = std::filesystem::path(filepath).stem().string() + "_suppressed.png";
  stbi_write_png(suppressed_file.c_str(), width, height, 1, h_suppressed, width * 1);
  printf("Written image as %s\n", suppressed_file.c_str());

// Relaxed cone map generation
	unsigned char *d_conemap;

  CUDA_CHECK(hipMalloc(&d_conemap, size * 4));
  
  // Launch kernel
  create_cone_map<<<blocks, threads>>>(d_input, d_derivative_image, d_suppressed, d_conemap, width, height, channels);
  CUDA_CHECK(hipDeviceSynchronize());
  
  // Copy result back
  unsigned char* h_conemap = (unsigned char*)malloc(size * 4);
  CUDA_CHECK(hipMemcpy(h_conemap, d_conemap, size * 4, hipMemcpyDeviceToHost));

  // Save image
	std::string conemap_file = std::filesystem::path(filepath).stem().string() + "_conemap.png";
  stbi_write_png(conemap_file.c_str(), width, height, 4, h_conemap, width * 4);
  printf("Written image as %s\n", conemap_file.c_str());

// Cleanup
  CUDA_CHECK(hipFree(d_input));
  CUDA_CHECK(hipFree(d_derivative_image));
  CUDA_CHECK(hipFree(d_dirs));
  CUDA_CHECK(hipFree(d_dirs_image));
  CUDA_CHECK(hipFree(d_fod));
  CUDA_CHECK(hipFree(d_second_derivative_image));
  CUDA_CHECK(hipFree(d_watershed));
  CUDA_CHECK(hipFree(d_suppressed));
  CUDA_CHECK(hipFree(d_conemap));
  stbi_image_free(h_input);
  free(h_derivative_image);
  free(h_dirs_image);
  free(h_second_derivative_image);
  free(h_watershed);
  free(h_suppressed);
  free(h_conemap);
}

int main(int argc, char** argv) {
	if (argc < 2) {
		fprintf(stderr, "No texture provided.\n");
		exit(0);
	}
	for (int i = 1; i < argc; ++i) {
		if (!std::filesystem::exists(argv[i]) && std::filesystem::is_regular_file(argv[i])) {
			fprintf(stderr,"No such file: %s\n", argv[i]);
		}
		convert_image(argv[i]);
	}

  return 0;
}
