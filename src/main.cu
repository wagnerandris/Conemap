#include "hip/hip_runtime.h"
// STD
#include <iostream>
#include <filesystem>
#include <string>
#include <vector>

// Boost
#include <boost/program_options.hpp>

#include "file_utils.cuh"
#include "kernels.cuh"

static std::filesystem::path output_path;

void convert_image(const char *filepath) {

  std::string output_name = output_path.append(std::filesystem::path(filepath).stem().string());

/* Load image */
  unsigned char *d_input_image = nullptr;
  int width, height, channels;
  if (!read_texture_to_device(d_input_image, filepath, &width, &height, &channels))
    return;

  size_t size = width * height;
  // size_t input_size = size * channels;

  // Threads/blocks
  // TODO what's optimal?
  dim3 threads(16, 16);
  dim3 blocks((width + threads.x - 1) / threads.x,
              (height + threads.y - 1) / threads.y);


/* First order derivatives */
  // Allocate device memory
  int *d_fod;
  unsigned char *d_fod_image, *d_fod_dirs, *d_fod_dirs_image;

  CUDA_CHECK(hipMalloc(&d_fod, size * 2 * sizeof(int)));
  CUDA_CHECK(hipMalloc(&d_fod_image, size * 3));
  CUDA_CHECK(hipMalloc(&d_fod_dirs, size));
  CUDA_CHECK(hipMalloc(&d_fod_dirs_image, size));

  // Launch kernel
  first_derivative<<<blocks, threads>>>(d_input_image, d_fod_image, d_fod_dirs,
                                        d_fod_dirs_image, d_fod, width, height,
                                        channels);
  CUDA_CHECK(hipDeviceSynchronize());

  // Write result image to file
  write_device_texture_to_file((output_name + "_fod.png").c_str(), d_fod_image,
                               width, height, 3);
  write_device_texture_to_file((output_name + "_fod_dirs.png").c_str(),
                               d_fod_dirs_image, width, height, 1);

/* Directional local maxima */
  // Allocate device memory
  unsigned char *d_local_max_dirs;
  CUDA_CHECK(hipMalloc(&d_local_max_dirs, size));

  // Launch kernel
  local_max_dirs<<<blocks, threads>>>(d_input_image, d_local_max_dirs, width,
                                      height, channels);
  CUDA_CHECK(hipDeviceSynchronize());

  // TODO local max dirs images


/* Relaxed cone map generation */
  // Allocate device memory
  unsigned char *d_cone_map;
  CUDA_CHECK(hipMalloc(&d_cone_map, size * 4));

  // Launch kernel
  create_cone_map2<<<blocks, threads>>>(d_input_image, d_fod_image,
                                        d_local_max_dirs, d_cone_map, width,
                                        height, channels);
  CUDA_CHECK(hipDeviceSynchronize());

  // Write result image to file
  write_device_texture_to_file((output_name + "_relaxed_cone_map.png").c_str(),
                               d_cone_map, width, height, 4);


// /* Second order derivatives and watershed */
//   // Allocate device memory
//   unsigned char *d_sod_image, *d_watershed;
//   CUDA_CHECK(hipMalloc(&d_sod_image, size * 4));
//   CUDA_CHECK(hipMalloc(&d_watershed, size));
//
//   // Launch kernel
//   second_derivative<<<blocks, threads>>>(d_fod, d_sod_image, d_watershed, width,
//                                          height);
//   CUDA_CHECK(hipDeviceSynchronize());
//
//   // Write result image to file
//   write_device_texture_to_file((output + "_sod.png").c_str(), d_sod_image,
//                                width, height, 4);
//   write_device_texture_to_file((output + "_watershed.png").c_str(), d_watershed,
//                                width, height, 1);
//
//   // Non maximum suppression
//   unsigned char *d_suppressed;
//   CUDA_CHECK(hipMalloc(&d_suppressed, size));
//
//   // Launch kernel
//   non_maximum_suppression<<<blocks, threads>>>(d_input_image, d_fod_dirs,
//                                                d_watershed, d_suppressed, width,
//                                                height, channels);
//   CUDA_CHECK(hipDeviceSynchronize());
//
//   // Write result image to file
//   write_device_texture_to_file((output + "_suppressed.png").c_str(),
//                                d_suppressed, width, height, 1);
//
//
// /* Relaxed cone map generation */
//   // Allocate device memory
//   unsigned char *d_cone_map;
//   CUDA_CHECK(hipMalloc(&d_cone_map, size * 4));
//
//   // Launch kernel
//   create_cone_map<<<blocks, threads>>>(d_input_image, d_fod_image, d_suppressed,
//                                        d_cone_map, width, height, channels);
//   CUDA_CHECK(hipDeviceSynchronize());
//
//   // Write result image to file
//   write_device_texture_to_file((output + "_relaxed_cone_map.png").c_str(),
//                                d_cone_map, width, height, 4);

/* Cleanup */
  CUDA_CHECK(hipFree(d_input_image));
  CUDA_CHECK(hipFree(d_fod_image));
  CUDA_CHECK(hipFree(d_fod_dirs));
  CUDA_CHECK(hipFree(d_fod_dirs_image));
  CUDA_CHECK(hipFree(d_fod));
  // CUDA_CHECK(hipFree(d_second_derivative_image));
  // CUDA_CHECK(hipFree(d_watershed));
  // CUDA_CHECK(hipFree(d_suppressed));
  CUDA_CHECK(hipFree(d_local_max_dirs));
  CUDA_CHECK(hipFree(d_cone_map));
}

int main(int argc, char* argv[]) {
  std::vector<std::string> input_files;

  // Possible options
  boost::program_options::options_description desc("Allowed options");
  desc.add_options()
    ("help,h", "produce help message")
    ("output,o", boost::program_options::value<std::filesystem::path>(&output_path)->default_value("."), "set path to output folder")
    ("input,i", boost::program_options::value<std::vector<std::string>>(&input_files), "input files");
		//TODO flip Y, depthmap, wrap

  // Positional options
  boost::program_options::positional_options_description pod;
  pod.add("input", -1);  // all remaining options

  boost::program_options::variables_map vm;

  try {
    boost::program_options::store(boost::program_options::command_line_parser(argc, argv).options(desc).positional(pod).run(), vm);
    boost::program_options::notify(vm);
  } catch (std::exception& e) {
    std::cerr << "Error: " << e.what() << "\n";
    return 1;
  }

	// Help
  if (vm.count("help")) {
    std::cout << "Usage: " << argv[0] << " [-o OUTPUT] INPUT [INPUT]...\n";
    std::cout << desc << "\n";
    return 0;
  }

	// No input
  if (input_files.empty()) {
    std::cerr << "Error: No input files provided.\n";
    return 1;
  }

  // Output
  if (!std::filesystem::exists(output_path)) {
		std::filesystem::create_directory(output_path);
  } else if (!std::filesystem::is_directory(output_path)) {
		std::cerr << "Error: " << output_path << " is not a directory.\n";
		return 1;
  }

	// OK
  for (auto file : input_files) {
    if (!std::filesystem::exists(file) || !std::filesystem::is_regular_file(file)) {
			std::cerr << "Error: " << file << " is not a file.\n";
			continue;
    }
    convert_image(file.c_str());
  }

  return 0;
}
